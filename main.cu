#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <chrono>

#include <fmt/core.h>

#include "jitify.hpp"

// UDF example
// Usage

int main() {
  using clk = std::chrono::system_clock;
  using ms = std::chrono::milliseconds;
  using us = std::chrono::microseconds;
  using ns = std::chrono::nanoseconds;
  using s = std::chrono::seconds;
  using time_point_t = decltype(clk::now());

  auto const program_source_1 = std::string{
      "my_program\n"
      "__global__\n"
      "void my_kernel(int *data, unsigned int num) {\n"
      "  unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;\n"
      "  if (id < num) {\n"
      "    int row = data[id];\n"
  };
  auto program_source_2 = std::string{};
  const auto program_source_3 = std::string{
      "    data[id] = row;\n"
      "  }\n"
      "}\n"
  };

  // read file into program_source_2
  auto file = std::fstream("UDF");
  auto buffer = std::stringstream{};
  buffer << file.rdbuf();
  program_source_2 = buffer.str();

  auto program_source = program_source_1 + program_source_2 + "\n" + program_source_3;
  fmt::print("[generated codes]\n{}", program_source);

  auto before = clk::now();
  static auto kernel_cache = jitify::JitCache{};
  auto program = kernel_cache.program(program_source);
  auto after = clk::now();
  auto duration = after - before;
  auto n_us = std::chrono::duration_cast<us>(duration).count();
  fmt::print("[compile time] {} us\n", n_us);

  int len = 20;
  int *h_data = new int[len];
  for (int i = 0; i < len; ++i) h_data[i] = i;

  fmt::print("[input items]\n");
  for (int i = 0; i < len; ++i) fmt::print("{} ", h_data[i]);
  fmt::print("\n");

  int *d_data;
  hipMalloc((void **) &d_data, sizeof(int) * len);
  hipMemcpy(d_data, h_data, sizeof(int) * len, hipMemcpyHostToDevice);
  dim3 grid(5);
  dim3 block(4);
  using jitify::reflection::type_of;
  program.kernel("my_kernel")
      .instantiate()
      .configure(grid, block)
      .launch(d_data, len);
  hipMemcpy(h_data, d_data, sizeof(int) * len, hipMemcpyDeviceToHost);
  hipFree(d_data);

  fmt::print("[output items]\n");
  for (int i = 0; i < len; ++i) fmt::print("{} ", h_data[i]);
  fmt::print("\n");

  delete[] h_data;

}
